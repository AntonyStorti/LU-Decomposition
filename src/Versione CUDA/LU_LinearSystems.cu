#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <time.h>
#include <sys/time.h>
#include "Solve_Systems.cu"

//Per assicurarmi di non eccedere il limite dei 1024 blocchi della "Tesla T4"
#define TILE 16


void salva_su_CSV(int N, double t, const char *filename);


int main(int argc, char** argv) {


    printf("\n+=========================================================================================+");
    printf("\n+====================| Decomposizione LU: Risoluzione di Sistemi Lineari |================+");
    printf("\n+=========================================================================================+\n\n");


    struct timeval start_real, end_real;
    double tempo_impiegato;

    //Dimensione della matrice (N x N)
    int n = atoi(argv[1]);


    //=========================== ALLOCAZIONE DINAMICA MEMORIA ========================//

    srand(100);

    //Matrice dei coefficienti
    double *a = (double*)malloc(n * n * sizeof(double));

    //Matrice Decomposta ( A = L*U )
    double *decomposta = (double*)malloc(n * n * sizeof(double));


    generaMatrice(a, n);


    //Vettore dei termini noti
    double *b = (double*)malloc(n * sizeof(double));

    //Popolamento del vettore dei termini noti:
    double min = -100;
    double max = 100;

    for (int i = 0; i < n; i++)
        b[i] = (double) rand() / RAND_MAX * (max - min);


    //=================================================================================//



    //=========================== ALLOCAZIONE MEMORIA SULLA GPU =======================//

    double *gpu_a;
    int numblock = n / TILE + ((n % TILE) ? 1 : 0);

    hipMalloc(&gpu_a, n * n * sizeof(double));
    hipMemcpy(gpu_a, a, n * n * sizeof(double), hipMemcpyHostToDevice);


    gettimeofday(&start_real, NULL);

    //##################################################################################//


                            LUDecompose(gpu_a, n, numblock);


    //##################################################################################//


    //Salvo la matrice decomposta sulla memoria della CPU
    hipMemcpy(decomposta, gpu_a, n * n * sizeof(double), hipMemcpyDeviceToHost);

    double** A = (double**)malloc(n * sizeof(double*));
    double** u = (double**)malloc(n * sizeof(double*));
    double** l = (double**)malloc(n * sizeof(double*));

    initialize_matrices(A, l, u, n);

    //Ottengo dalla matrice decomposta: L ed U
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            A[i][j] = decomposta[i * n + j];
        }
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                if (i >= k)
                    l[i][k] = A[i][k];
                else
                    l[i][k] = 0;

                if (k == j)
                    u[k][j] = 1;
                else if (k < j)
                    u[k][j] = A[k][j];
                else
                    u[k][j] = 0.0;
            }
        }
    }


    //##################################################################################//


                            int core_usati = LUSolve(n, l, u, b);


    //##################################################################################//


    gettimeofday(&end_real, NULL);
    tempo_impiegato = (double)(end_real.tv_sec - start_real.tv_sec) +
                      (double)(end_real.tv_usec - start_real.tv_usec) / 1000000.0;


    // Stampa risultati
    printf("\n\nRisultato del sistema di equazioni:\n\n");
    for (int i = 0; i < n; i++)
        printf("x%d = %f\n", i + 1, b[i]);

    printf("\n\nLa CPU remota ha disponibili: %d cores\n", core_usati);
    printf("Sono stati usati per la parallelizzazione: %d cores !\n", core_usati);

    printf("\n\n=========================================");
    printf("\nIl Tempo impiegato per risolvere il sistema è: %lf\n", tempo_impiegato);
    printf("\n\n\n");


    salva_su_CSV(n, tempo_impiegato, "CUDA.csv");


    hipFree(gpu_a);
    free(a);
    free(decomposta);
    free(b);
    deallocate_matrices(A, l, u, n);



    return 0;

}





void salva_su_CSV(int N, double t, const char *filename){


    FILE *file = fopen(filename, "a");

    if (file != NULL) {

        // Determina se il file è appena stato creato
        long pos = ftell(file);

        if (pos == 0) {
            // Il file è vuoto, quindi scrivo l'intestazione
            fprintf(file, "Dimensione Matrice (NxN) ; Tempo di Esecuzione (sec)\n");
        }

        fprintf(file, "%d;%f\n", N, t);

        fclose(file);

    }   else {
        perror("\nErrore nel salvataggio dei dati sul CSV !\n");
    }


}